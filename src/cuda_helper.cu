#include "cuda_helper.h"

#include <assert.h>
#include <time.h>
#include <iostream>
#include <complex>
#include <fstream>
#include <vector>
#include <omp.h>

namespace refft {

#define CUDA_ERROR_CHECK
#define CUDA_CALL(x)                                                 \
  do {                                                               \
    if ((x) != hipSuccess) {                                        \
      printf("Error at %s:%d\n", __FILE__, __LINE__);                \
      printf("error: %s\n", hipGetErrorString(hipGetLastError())); \
      abort();                                                       \
    }                                                                \
  } while (0)

void __cudaCheckError(const char* file, const int line) {
  hipError_t err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    std::cerr << "cudaCheckError failed at " << file << " " << line << "\n";
    std::cerr << "error: " << hipGetErrorString(err) << "\n";
    exit(-1);
  }
  return;
}

void CudaHostSync() { hipDeviceSynchronize(); }

void* DeviceMalloc(size_t len) {
  void* data;
  CUDA_CALL(hipMalloc(&data, len));
  return data;
}

void* DeviceMalloc(const std::vector<std::complex<float>> input) {
  void* dst = DeviceMalloc(input.size() * sizeof(std::complex<float>));
  CUDA_CALL(hipMemcpy(dst, input.data(), input.size() * sizeof(std::complex<float>),
                       hipMemcpyHostToDevice));
  return dst;
}

std::vector<std::complex<float>> D2H(const std::complex<float>* input, size_t elems) {
  std::vector<std::complex<float>> host(elems);
  CUDA_CALL(hipMemcpy(host.data(), input, elems * sizeof(std::complex<float>),
                       hipMemcpyDeviceToHost));
  return host;
}


void DeviceFree(uint32_t* p) { CUDA_CALL(hipFree((void*)p)); }
void DeviceFree(bool* p) { CUDA_CALL(hipFree((void*)p)); }
void DeviceFree(std::complex<float>* p) { CUDA_CALL(hipFree((void*)p)); }

}  // namespace cucrt
