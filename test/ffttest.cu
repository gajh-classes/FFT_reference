#include "hip/hip_runtime.h"
#define MAX(X, Y) ((X) > (Y) ? (X) : (Y))
#include "cuda_helper.h"
#include "ffthelper.h"
#include "utils.h"

//#include "gtest/gtest.h"

#include <algorithm>
#include <chrono>
#include <cmath>
#include <complex>
#include <fstream>
#include <iostream>
#include <random>
#include <vector>

using HostVec = std::vector<uint32_t>;
using ComplexVec = std::vector<std::complex<float>>;
using refft::utils::genComp;
using refft::utils::genZero;

void ReadFile(ComplexVec a, std::string file_name) {
  std::ifstream input;
  input.open(file_name);
  if (!input.good()) {
    std::cout << "Could not open " << file_name << std::endl;
    std::exit(0);
  }
  float real;
  float imag;
  for (int i = 0; i < a.size(); i++) {
    input >> real;
    input >> imag;
    a[i] = {real, imag};
  }
  input.close();
}

int main(int argc, char **argv){
  refft::blocksize = std::stoi(argv[1]);
  refft::gridsize = std::stoi(argv[2]);
  int N = 32768; 
  for(int iter = 0; iter < 1; iter++){
    ComplexVec h_a(N);
    ComplexVec res_ref(N);
    ReadFile(h_a, "Polynomial_Coeff.txt");
    ReadFile(res_ref, "Output_Coeff.txt");
    std::complex<float>* d_alpha =
        (std::complex<float>*)refft::DeviceMalloc(h_a);

    auto start = clock();
    refft::FftHelper::ExecStudentFft(d_alpha, N);
    refft::CudaHostSync();
    auto end = clock();
    double elapsed_sec = double(end - start) / CLOCKS_PER_SEC;

    ComplexVec res = refft::D2H(d_alpha, N);
    for (unsigned int i = 0; i < res.size(); i++) {
      if (!(abs(res_ref[i].real() - res[i].real()) < 0.001)) {
        std::cout << "Wrong value in index " << i << std::endl;
        std::cout << "Reference : " << res_ref[i].real() << std::endl;
        std::cout << "Calculated : " << res[i].real() << std::endl;
        std::exit(0);
      }
      if (!(abs(res_ref[i].imag() - res[i].imag()) < 0.001)) {
        std::cout << "Wrong value in index " << i << std::endl;
        std::cout << "Reference : " << res_ref[i].imag() << std::endl;
        std::cout << "Calculated : " << res_ref[i].imag() << std::endl;
        std::exit(0);
      }
    }
    refft::DeviceFree(d_alpha);
    std::cout << "Duration of StudentFft : " << elapsed_sec << "(sec)"
              << std::endl;
  }
  return 0;
}
